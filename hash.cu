#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>

#define ROTR(x,n) (((x) >> (n)) | ((x) << ((32) - (n))))

__constant__ uint32_t cst_k[] = {
   0x428a2f98, 0x71374491, 0xb5c0fbcf, 0xe9b5dba5, 0x3956c25b, 0x59f111f1, 0x923f82a4, 0xab1c5ed5,
   0xd807aa98, 0x12835b01, 0x243185be, 0x550c7dc3, 0x72be5d74, 0x80deb1fe, 0x9bdc06a7, 0xc19bf174,
   0xe49b69c1, 0xefbe4786, 0x0fc19dc6, 0x240ca1cc, 0x2de92c6f, 0x4a7484aa, 0x5cb0a9dc, 0x76f988da,
   0x983e5152, 0xa831c66d, 0xb00327c8, 0xbf597fc7, 0xc6e00bf3, 0xd5a79147, 0x06ca6351, 0x14292967,
   0x27b70a85, 0x2e1b2138, 0x4d2c6dfc, 0x53380d13, 0x650a7354, 0x766a0abb, 0x81c2c92e, 0x92722c85,
   0xa2bfe8a1, 0xa81a664b, 0xc24b8b70, 0xc76c51a3, 0xd192e819, 0xd6990624, 0xf40e3585, 0x106aa070,
   0x19a4c116, 0x1e376c08, 0x2748774c, 0x34b0bcb5, 0x391c0cb3, 0x4ed8aa4a, 0x5b9cca4f, 0x682e6ff3,
   0x748f82ee, 0x78a5636f, 0x84c87814, 0x8cc70208, 0x90befffa, 0xa4506ceb, 0xbef9a3f7, 0xc67178f2
};

__device__
void sha256(uint64_t N, char *array, uint32_t *w, uint32_t *h_result, int nonce_position, uint64_t nonce_value)
{
    /* N is size of array
       array is preprocessed data to hash
       k is the 64 variables constants to use in sha256
       w is a 64 entry message schedule array; it can contain anything,
               it will be written over. This prevents mallocs in hash function.
       h is an 8 entry array where the hash result will be put
    */
    h_result[0] = 0x6a09e667;
    h_result[0] = 0xbb67ae85;
    h_result[0] = 0x3c6ef372;
    h_result[0] = 0xa54ff53a;
    h_result[0] = 0x510e527f;
    h_result[0] = 0x9b05688c;
    h_result[0] = 0x1f83d9ab;
    h_result[0] = 0x5be0cd19;

    /* pass through array in 512bits chunks (64 bytes) */
    int N_chunks = N>>6;
    for (int chunk_i = 0; chunk_i < N_chunks; chunk_i++) {
        int chunk_start = chunk_i<<6;
        
        /* copy chunk in first 8 values of w */
        memcpy(w, &array[chunk_start], 64);
        
        // TODO : review this chunk of code
        // calculate pos relative to chunk_start in terms of bytes
        int pos_in_w = nonce_position - chunk_start;
        //change nonce if it is in the chunk
        if (0 <= pos_in_w <= 64) {
            if (pos_in_w <= 56) {
                /* if the nonce is completely in the chunk,
                   copy it completely at the right location
                   (the syntax is weird because nonce_position is in bytes
                    while w is uint32_t*) */
                *(w + pos_in_w) = nonce_value;
            } else {
                // copy only a left part of the nonce
                memcpy(w + pos_in_w,
                       &nonce_value,
                       64-pos_in_w);
            }
        }
        if (-8 < pos_in_w < 0) {
            // copy only a right part of the nonce
            memcpy(w + pos_in_w,
                   &nonce_value + 8 + pos_in_w,
                   8 + pos_in_w);
        }

        /* complete w by following some weird rules */
        for (int i = 16; i < 64; i++) {
            uint32_t w15 = w[i-15];
            uint32_t s0 = ROTR(w15, 7) xor ROTR(w15, 18) xor w15 << 3;
            uint32_t w2 = w[i-2];
            uint32_t s1 = ROTR(w2, 17) xor ROTR(w2, 19) xor w2 << 10;
            w[i] = w[i-16] + s0 + w[i-7] + s1;
        }

        uint32_t a = h_result[0];
        uint32_t b = h_result[1];
        uint32_t c = h_result[2];
        uint32_t d = h_result[3];
        uint32_t e = h_result[4];
        uint32_t f = h_result[5];
        uint32_t g = h_result[6];
        uint32_t h = h_result[7];

        for (int i = 0; i < 64; i++) {
            uint32_t S1 = ROTR(e, 6) xor ROTR(e, 11) xor ROTR(e, 25);
            uint32_t ch = (e and f) xor ((not e) and g);
            uint32_t temp1 = h + S1 + ch + cst_k[i] + w[i];
            uint32_t S0 = ROTR(a, 2) xor ROTR(a, 13) xor ROTR(a, 22);
            uint32_t maj = (a and b) xor (a and c) xor (b and c);
            uint32_t temp2 = S0 + maj;

            h = g;
            g = f;
            f = e;
            e = d + temp1;
            d = c;
            c = b;
            b = a;
            a = temp1 + temp2;
        }
        h_result[0] += a;
        h_result[1] += b;
        h_result[2] += c;
        h_result[3] += d;
        h_result[4] += e;
        h_result[5] += f;
        h_result[6] += g;
        h_result[7] += h;
    }
}


char *preprocess_sha256(uint64_t length, char *array)
{
    // final_length is smallest number over (or equal) to length+1+8 that is divisible by 512
    uint64_t final_length = ((length + 8)>>9 +1)<<9;

    // allocate space on ram
    char *host_array;
    host_array = (char*) malloc(final_length);
    // initialize final padding at 0
    for (int i = 1; i <= 64; i += 1) {
        host_array[final_length - i] = 0;
    }

    // allocate memory on gpu
    char *device_array = 0;
    hipMalloc((void**) &device_array, final_length);

    // create processed array
    // copy message
    memcpy(host_array, array, length);
    // put a 1 after the message
    host_array[length] = '\x80';
    // write message length at the end
    char last_char = 0;
    for (int shift = 54; shift >= 0; shift -= 8) {
        last_char = length>>shift - last_char<<8;
        host_array[final_length - 8] = last_char;
    }


    // copy processed array to the device
    hipMemcpy(device_array, array, final_length, hipMemcpyHostToDevice);

    // free host memory space
    free(host_array);

    return device_array;
}


uint32_t *prepare_h_results(int n_of_threads)
{
    uint32_t *h_results_pos;
    // 32 bytes for each tread (256bits)
    hipMalloc((void**) &h_results_pos, n_of_threads * 32);
    return h_results_pos;
}


uint32_t *prepare_working_memories(int n_of_threads)
{
    uint32_t *working_memories;
    // 64 word long working memory for each thread
    hipMalloc((void**) &working_memories, n_of_threads * sizeof(uint32_t) * 64);
    return working_memories;
}

/*
   Try hashing with a few different nonces,
   if it has found a hash smaller than target
   it returns the corresponding nonce.
   Otherwise it returns 0 (the nonce 0 is never tested)
*/
__device__
uint64_t hash_range(uint64_t N, char *array, uint32_t *w, uint32_t *h_result, int nonce_position, uint32_t *target)
{
    for (uint64_t nonce = 1; nonce != 0; nonce ++) {
        sha256(N, array, w, h_result, nonce_position, nonce);
        if target_reached(h_result, target) {
            return nonce
        }
        nonce ++;
    }
    return 0
}

__device__
bool target_reached(uint32_t h_result, uint32_t target)
{
    for (int i = 0; i<8; i++) {
        if(h_result[i] > target[i]) {
            return false;
        } else if (h_result[i] < target[i]) {
            return true;
        }
    }
    /* if h_result == target, consider the target reached */
    return true;
}

__global__
void launch_hash_range(uint64_t N, char *array, uint32_t *w, uint32_t *h_result, int nonce_position, uint32_t *target)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    h_result = h_result + (8*sizeof(uint32_t)) * tid;
    w = w + (64*sizeof(uint32_t)) * tid;
    /* TODO : modify a base of the nonce and shift the nonce_position to ensure it is on a fitting address for uint64_t copies */
    hash_range(N, array, w, h_result, nonce_position, target);
}

uint64_t find_solution(char *text, uint64_t text_size)
{
    int n_of_threads = 2;
    char *device_text = preprocess_sha256(text_size, text);
    uint32_t *h_results = prepare_h_results(n_of_threads);
    uint32_t *working_memories = prepare_working_memories(n_of_threads);
    /* TODO : launch a bunch of launch_hash_range 
              and find a way to get resulting nonce 
              and close every thread when done*/
}


uint32_t *hash_once(char *text, uint64_t text_size)
{
    int n_of_threads = 1;
    char *device_text = preprocess_sha256(text_size, text);
    uint32_t *h_results = prepare_h_results(n_of_threads);
    uint32_t *working_memories = prepare_working_memories(n_of_threads);
}


/* for test purposes */
int main(int argc, char *argv[])
{
    char *text;
    uint64_t text_size = 0;
    if (argc > 1) {
        text_size = strlen(argv[1]);
        text = strdup(argv[1]);
    }else {
        text = strdup("");
    }
    preprocess_sha256(text_size, text);

    return EXIT_SUCCESS;
}
